#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

// Device code
__global__ void MyKernel(int* devPtr, size_t pitch)
{
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int* row = (int*)((char*)devPtr + ty * pitch) + tx;
    *row *= 2;
}


int main()
{
    // Host code
    int* devPtr;
    size_t pitch;
    hipError_t err; 
    err = hipMallocPitch(&devPtr, &pitch,
            width * sizeof(float), height);
    std::cout << pitch << std::endl;
    if(err != hipSuccess){
        fprintf(stderr, "%s", hipGetErrorString(err));
    }else{
        std::cout << "coppied successfully" <<std::endl;
    }
    err = hipMemcpy2D(devPtr, pitch, &M, width*sizeof(int), width*sizeof(int), height, hipMemcpyDefault); 
    if(err != hipSuccess){
        fprintf(stderr, "%s", hipGetErrorString(err));
    }else{
        std::cout << "coppied successfully" <<std::endl;
    }
    dim3 block_dim(20,20);
    MyKernel<<<1, block_dim>>>(devPtr, pitch);

    err = hipMemcpy2D(&C, width*sizeof(int), devPtr, pitch, width*sizeof(int), height, hipMemcpyDeviceToHost); 
    if(err != hipSuccess){
        fprintf(stderr, "%s", hipGetErrorString(err));
    }
    print_arr(C);

}
